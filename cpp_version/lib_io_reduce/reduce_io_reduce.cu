#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <iostream>
#include "reduce_io_reduce.h"
#include "dna_io_reduce.h"

__global__ void reduceKernel(int *d_in, int* d_out, int totalsize) {

	int myID = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = threadIdx.x;

	extern __shared__ float sdata[];

	if (myID < totalsize) {
        sdata[tid] = d_in[myID];
	}
	else {
		sdata[tid] = 0;
	}

	__syncthreads();

	if (myID >= totalsize) {
		if (tid == 0) {
			d_out[blockIdx.x] = 0;
		}
		return;
	}
    
	//divide threads into two parts according to threadID, and add the right part to the left one, lead to reducing half elements, called an iteration; iterate until left only one element
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] = sdata[tid + s] + sdata[tid];
		}
		__syncthreads(); //ensure all adds at one iteration are done
	}

	if (tid == 0) {
		d_out[blockIdx.x] = sdata[0];
	}
}

__global__ void initialKernel(uint8_t* d_input, int* d_output, int totalsize, int toReduce){

	int myID = threadIdx.x + blockIdx.x * blockDim.x;

	if (myID < totalsize) {
        if (d_input[myID] == toReduce){
		    d_output[myID] = 1;
        }  
        else{
            d_output[myID] = 0;
        }
	}

};

void reduction(uint8_t* d_in, int* result, int input_size, int toReduce) {
	int blocksize = 64;

    int* d_raw;
    hipMalloc(&d_raw, input_size * sizeof(int));
	
    initialKernel<<<roundup(input_size, blocksize),blocksize>>>(d_in, d_raw, input_size, toReduce);

	const int shared_mem_size = sizeof(int)*blocksize;
	int currentSize = input_size;
	int tsize = input_size;
	while (1) {
		int* d_current_out;
		tsize = currentSize;
		currentSize = roundup(tsize, blocksize);
		// std::cout << currentSize << std::endl;
		hipMalloc(&d_current_out, currentSize * sizeof(int));

		reduceKernel <<< currentSize, blocksize, shared_mem_size >>> (d_raw, d_current_out, tsize);
		hipFree(d_raw);

		d_raw = d_current_out;

		if (currentSize == 1)
		{
			break;
		}
	}

	hipMemcpy(result, d_raw, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_raw);
}
